#include "hip/hip_runtime.h"

#include <iostream>
#include "off_vector.cuh"

__global__ void kernel(OffVector M){

    for (int i = 0; i < 11; i++){
        printf("%f\n", M[i]);
    }
}


int main() {
    std::cout << "Starting" << std::endl;
    bool finished = false;

    int max_capability = 10;
    OffVector M(max_capability, "matrix.bin");

    // Create CUDA streams
    hipStream_t kernelStream, loadStream;
    hipStreamCreate(&kernelStream);
    hipStreamCreate(&loadStream);

    // Create event to monitor kernel completion
    hipEvent_t kernelDoneEvent;
    hipEventCreate(&kernelDoneEvent);

    // Launch kernel in kernelStream
    kernel<<<1, 1, 0, kernelStream>>>(M);
    hipEventRecord(kernelDoneEvent, kernelStream); // Record completion event
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        std::cerr << "Kernel launch error: " << hipGetErrorString(error) << std::endl;
        return -1;
    }

    while (!finished) {
        std::cout << "Waiting for event..." << std::endl;
        
        // Check if kernel has finished
        hipError_t status = hipStreamQuery(kernelStream);
        if (status == hipSuccess) {
            finished = true;
        } else if (status == hipErrorNotReady) {
            bool need_load;
            hipMemcpyAsync(&need_load, M.d_need_load, sizeof(bool), hipMemcpyDeviceToHost, loadStream);
            hipStreamSynchronize(loadStream);
            if (need_load) {
                M.check_and_load(loadStream);
                hipStreamSynchronize(loadStream);
            }
        } else {
            std::cerr << "Error in kernel execution: " << hipGetErrorString(status) << std::endl;
            finished = true;
        }
    }

    // Cleanup
    hipStreamDestroy(kernelStream);
    hipStreamDestroy(loadStream);
    hipEventDestroy(kernelDoneEvent);

    return 0;
}