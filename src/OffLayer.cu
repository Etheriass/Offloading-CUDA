#include "OffLayer.cuh"
#include <iostream>
#include <fstream>
#include <vector>

OffLayer::OffLayer(const std::string &filename, size_t total_layers, const std::vector<size_t> &layer_rows, const std::vector<size_t> &layer_cols)
    : total_layers(total_layers), layer_rows(layer_rows), layer_cols(layer_cols), current_layer(1)
{
    file.open(filename, std::ios::binary);
    if (!file.is_open())
    {
        throw std::runtime_error("Failed to open file: " + filename);
    }
    hipMalloc(&device_matrix, getMaxLayerSize() * sizeof(float));
}

OffLayer::~OffLayer()
{
    hipFree(device_matrix);
    file.close();
}

float* OffLayer::getLayer(size_t layer_index)
{
    if (layer_index != current_layer)
    {
        loadLayer(layer_index);
        current_layer = layer_index;
    }
    return device_matrix;
}

void OffLayer::loadLayer(size_t layer_index)
{
    size_t offset = 0;
    for (size_t i = 0; i < layer_index; ++i)
    {
        offset += layer_rows[i] * layer_cols[i] * sizeof(float);
    }
    file.seekg(offset, std::ios::beg);
    if (file.fail())
    {
        throw std::runtime_error("Failed to seek to position in file");
    }
    size_t elements_to_read = layer_rows[layer_index] * layer_cols[layer_index];
    std::vector<float> host_matrix(elements_to_read);
    file.read(reinterpret_cast<char *>(host_matrix.data()), elements_to_read * sizeof(float));
    hipMemcpy(device_matrix, host_matrix.data(), elements_to_read * sizeof(float), hipMemcpyHostToDevice);
}

size_t OffLayer::getLayerRows(size_t layer_index)
{
    return layer_rows[layer_index];
}

size_t OffLayer::getLayerCols(size_t layer_index)
{
    return layer_cols[layer_index];
}

size_t OffLayer::getMaxLayerSize()
{
    size_t max_size = 0;
    for (size_t i = 0; i < total_layers; ++i)
    {
        size_t size = layer_rows[i] * layer_cols[i];
        if (size > max_size)
        {
            max_size = size;
        }
    }
    return max_size;
}