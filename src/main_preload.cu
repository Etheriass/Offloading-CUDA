#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <vector>
#include <chrono>
// #include "OffLayer.cuh"

class OffLayer {
private:
    size_t total_layers;
    std::vector<size_t> layer_rows;
    std::vector<size_t> layer_cols;
    std::ifstream file;
    float* device_matrix;
    size_t current_layer;

public:
    OffLayer(const std::string& filename, size_t total_layers, const std::vector<size_t>& layer_rows, const std::vector<size_t>& layer_cols)
        : total_layers(total_layers), layer_rows(layer_rows), layer_cols(layer_cols), current_layer(1) {
        file.open(filename, std::ios::binary);
        if (!file.is_open()){
            throw std::runtime_error("Failed to open file: " + filename);
        }
        hipMalloc(&device_matrix, getMaxLayerSize() * sizeof(float));
    }

    ~OffLayer() {
        hipFree(device_matrix);
        file.close();
    }

    float* getLayer(size_t layer_index) {
        if (layer_index != current_layer) {
            current_layer = layer_index;
        }
        return device_matrix;
    }

    void preloadLayer(size_t layer_index, std::vector<float>& host_matrix) {
        size_t offset = 0;
        for (size_t i = 0; i < layer_index; ++i) {
            offset += layer_rows[i] * layer_cols[i] * sizeof(float);
        }
        file.seekg(offset, std::ios::beg);
        if (file.fail()){
            throw std::runtime_error("Failed to seek to position in file");
        }
        size_t elements_to_read = layer_rows[layer_index] * layer_cols[layer_index];
        host_matrix.resize(elements_to_read);
        file.read(reinterpret_cast<char*>(host_matrix.data()), elements_to_read * sizeof(float));
    }

    void loadPreloadedLayer(const std::vector<float>& host_matrix) {
        hipMemcpy(device_matrix, host_matrix.data(), host_matrix.size() * sizeof(float), hipMemcpyHostToDevice);
    }

    size_t getLayerRows(size_t layer_index) {
        return layer_rows[layer_index];
    }

    size_t getLayerCols(size_t layer_index) {
        return layer_cols[layer_index];
    }

    size_t getMaxLayerSize() {
        size_t max_size = 0;
        for (size_t i = 0; i < total_layers; ++i) {
            size_t size = layer_rows[i] * layer_cols[i];
            if (size > max_size) {
                max_size = size;
            }
        }
        return max_size;
    }
};

/* Kernel to multiply a matrix with a vector */
__global__ void matVecMul(float* matrix, float* vector, float* result, size_t rows, size_t cols) {
    size_t row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < rows) {
        float sum = 0.0f;
        for (size_t col = 0; col < cols; ++col) {
            sum += matrix[row * cols + col] * vector[col];
        }
        result[row] = sum;
    }
}

/* Function to run inference on a set of layers stored in a file */
void runInference(const std::string &filename, size_t total_layers, std::vector<size_t> layer_rows, std::vector<size_t> layer_cols) {
    auto start = std::chrono::high_resolution_clock::now();

    OffLayer matrices(filename, total_layers, layer_rows, layer_cols);

    // Build a random input vector for first layer
    std::vector<float> host_vector(layer_cols[0]);
    srand(static_cast<unsigned>(time(0)));
    for (size_t i = 0; i < layer_cols[0]; ++i) {
        host_vector[i] = static_cast<float>(rand()) / RAND_MAX;
    }

    float* device_vector;
    float* device_result;
    hipMalloc(&device_vector, layer_cols[0] * sizeof(float));
    hipMalloc(&device_result, matrices.getMaxLayerSize() * sizeof(float));
    hipMemcpy(device_vector, host_vector.data(), layer_cols[0] * sizeof(float), hipMemcpyHostToDevice);

    std::vector<float> preloaded_matrix;
    matrices.preloadLayer(0, preloaded_matrix);

    // Sequentially multiply with each layer
    for (size_t layer_index = 0; layer_index < total_layers; ++layer_index) {
        size_t rows = matrices.getLayerRows(layer_index);
        size_t cols = matrices.getLayerCols(layer_index);

        // Load the layer matrix
        auto start_loading_layer = std::chrono::high_resolution_clock::now();
        matrices.loadPreloadedLayer(preloaded_matrix);
        auto end_loading_layer = std::chrono::high_resolution_clock::now();
        auto duration = std::chrono::duration_cast<std::chrono::microseconds>(end_loading_layer - start_loading_layer);
        std::cout << "Loading layer " << layer_index << " took: " << duration.count() << " us" << std::endl;

        if (layer_index + 1 < total_layers) {
            matrices.preloadLayer(layer_index + 1, preloaded_matrix);
        }

        size_t threads_per_block = 256;
        size_t blocks_per_grid = (rows + threads_per_block - 1) / threads_per_block;

        // Multiply the matrix with the input vector
        auto start_layer_multiplication = std::chrono::high_resolution_clock::now();
        matVecMul<<<blocks_per_grid, threads_per_block>>>(matrices.getLayer(layer_index), device_vector, device_result, rows, cols);
        hipDeviceSynchronize();
        auto end_layer_multiplication = std::chrono::high_resolution_clock::now();
        auto duration_layer_multiplication = std::chrono::duration_cast<std::chrono::microseconds>(end_layer_multiplication - start_layer_multiplication);
        std::cout << "Multiplying layer " << layer_index << " took: " << duration_layer_multiplication.count() << " us" << std::endl;

        // Update the input vector for the next iteration
        hipFree(device_vector);
        hipMalloc(&device_vector, rows * sizeof(float));
        hipMemcpy(device_vector, device_result, rows * sizeof(float), hipMemcpyDeviceToDevice);

        if (layer_index == total_layers - 1) {
            host_vector.resize(rows);
        }
    }

    hipMemcpy(host_vector.data(), device_result, host_vector.size() * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(device_vector);
    hipFree(device_result);

    // std::cout << "Final output vector:" << std::endl;
    // for (size_t i = 0; i < host_vector.size(); ++i) {
    //     std::cout << host_vector[i] << " ";
    // }
    // std::cout << std::endl;

    auto end = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start);
    std::cout << "Inference took: " << duration.count() << " ms" << std::endl;
}

int main(int argc, char** argv) {
    if (argc != 3) {
        std::cerr << "Usage: " << argv[0] << " <filename> <total_layers>" << std::endl;
        return 1;
    }

    const std::string &filename = argv[1];
    size_t total_layers = std::stoi(argv[2]);
    std::vector<size_t> layer_rows = {10000, 10000, 10000, 10000, 10000, 10000, 10000, 10000, 10000, 10000};
    std::vector<size_t> layer_cols = {10000, 10000, 10000, 10000, 10000, 10000, 10000, 10000, 10000, 10000};

    runInference(filename, total_layers, layer_rows, layer_cols);
    return 0;
}
